
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cstdio>


__constant__ int device_n;


__global__
void add(int n, float* x, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;
    
    if (threadIdx.x == 0) {
        printf("%d %d %d\n", blockIdx.x, gridDim.x, blockDim.x);
    }

	for (int i = index; i < n; i += stride) {
		y[i] = x[i] + y[i];
	}	
}


int main() {
	int N = 1 << 20;
	size_t size = N * sizeof(float);
	float *x = (float*)malloc(size);
	float *y = (float*)malloc(size);

	float *d_x, *d_y;

	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);


	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}


	hipMemcpy(x, d_x, size, hipMemcpyHostToDevice);
	hipMemcpy(y, d_y, size, hipMemcpyHostToDevice);

	int blockSize = 256;

	int numBlocks = (N + blockSize - 1) / blockSize;

	add<<<numBlocks / 2, blockSize>>>(N, d_x, d_y);

	hipDeviceSynchronize();	
	hipMemcpy(d_y, y, size, hipMemcpyDeviceToHost);

	float maxError = 0.0f;
	for (int i = 0; i < N; i++) {
		maxError = fmax(maxError, fabs(y[i]-3.0f));
	}
	std::cout << "Max error: " << maxError << std::endl;

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
	return 0;
}
