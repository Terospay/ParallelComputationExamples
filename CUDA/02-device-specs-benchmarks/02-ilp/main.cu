
#include <hip/hip_runtime.h>
__global__ void SumV0(int* x, int* y, int* result) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    int stride = gridDim.x * blockDim.x;

    result[tid] = x[tid] + y[tid];

}

__global__ void SumV1(int *x, int* y, int* result) {
    int double_tid = threadIdx.x + 2 * blockDim.x * blockIdx.x;

    result[double_tid] = x[double_tid] + y[double_tid];
    result[double_tid + blockDim.x] = x[double_tid + blockDim.x] + y[double_tid + blockDim.x]; 
}

int main() {
    int array_size = 1 << 26;
    
    int *h_x = new int[array_size];
    int *h_y = new int[array_size];
    
    for (int i = 0; i < array_size; ++i) {
        h_x[i] = i;
        h_y[i] = 2 * i;
    }

    int* d_x;
    int* d_y;
    int* d_result;
    
    int num_bytes = sizeof(*h_x) * array_size;
    hipMalloc(&d_x, num_bytes);
    hipMalloc(&d_y, num_bytes);
    hipMalloc(&d_result, num_bytes);

    hipMemcpy(d_x, h_x, num_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, num_bytes, hipMemcpyHostToDevice);

    int block_size = 512;

    int num_blocks = (array_size + block_size - 1) / block_size;

    SumV1<<<num_blocks / 2, block_size>>>(d_x, d_y, d_result);
    SumV0<<<num_blocks, block_size>>>(d_x, d_y, d_result);


    int *h_result = new int[array_size];

    hipMemcpy(h_result, d_result, num_bytes, hipMemcpyDeviceToHost);
   
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    delete[] h_x;
    delete[] h_y;
    delete[] h_result;
    return 0;
}
