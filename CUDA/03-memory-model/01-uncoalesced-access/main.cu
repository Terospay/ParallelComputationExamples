
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

__global__
void add(int n, float* x, float* y, float* z) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride) {
        int iindex = (i / 32) * 32 + (i) % 32; // call for coalescing 
		z[i] = 2.0f * x[iindex] + y[iindex];
	}	
}


int main() {
	int N = 1 << 28;
	size_t size = N * sizeof(float);
	float *x = (float*)malloc(size);
	float *y = (float*)malloc(size);
    float *z = (float*)malloc(size);

	float *d_x, *d_y, *d_z;

	hipMalloc(&d_x, size);
	hipMalloc(&d_y, size);
    hipMalloc(&d_z, size);


	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}


	hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

	int blockSize = 256;

	int numBlocks = (N + blockSize - 1) / blockSize;

    hipEvent_t start;
    hipEvent_t stop;

    // Creating event
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
	add<<<numBlocks, blockSize>>>(N, d_x, d_y, d_z);

    hipEventRecord(stop);

	hipMemcpy(z, d_z, size, hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    
    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << milliseconds << " elapsed" << std::endl;

	hipFree(d_x);
	hipFree(d_y);
    hipFree(d_z);
	free(x);
	free(y);
    free(z);
	return 0;
}
