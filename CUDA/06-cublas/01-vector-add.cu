
#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    hipError_t cuda_stat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int array_size = (1 << 22);
    float* h_x = new float[array_size];

    for (int i = 0; i < array_size; ++i) {
        h_x[i] = i * 1.0f;
    }

    float* d_x;

    cuda_stat = hipMalloc(&d_x, sizeof(float) * array_size);

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(
        array_size,
        sizeof(*h_x),
        h_x,
        /* space by host */ 1,
        d_x,
        /* space by device */ 1
    );
    int result;

    stat = hipblasIsamax(handle, array_size, d_x, 3, &result);

    // 0, 3, 6, 9, ...
    // 1, 2, 3, 4, 5, ...
    // max element located at (1398102 - 1) * 3

    std::cout << result << std::endl;

    return 0;
}