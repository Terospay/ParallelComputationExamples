// #include <cstdio>

#include <hip/hip_runtime.h>
#include <iostream>
#include <hipblas.h>

int main() {
    hipError_t cuda_stat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;

    int array_size = (1 << 22);
    float* h_x = new float[array_size];

    for (int i = 0; i < array_size; ++i) {
        h_x[i] = i * 1.0f;
    }

    float* d_x;

    cuda_stat = hipMalloc(&d_x, sizeof(float) * array_size);

    stat = hipblasCreate(&handle);
    stat = hipblasSetVector(array_size, sizeof(*h_x), h_x, /* space by host */ 1, d_x, /* space by device */ 1);
    int result;

    stat = hipblasIsamax(handle, array_size, d_x, 1, &result);

    std::cout << result << std::endl;

    return 0;
}